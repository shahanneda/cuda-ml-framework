#include <iostream>
#include <hip/hip_runtime.h>

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    std::cout << "CPU: Hello World!" << std::endl;
    return 0;
}