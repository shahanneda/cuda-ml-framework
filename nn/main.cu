#include "shape.h"
#include <iostream>
#include <stdio.h>
#include "CudaException.h"

using namespace std;

int main(void)
{
    Shape shape = Shape(100, 200);
    cout << "shape x: " << shape.x << ", shape y: " << shape.y << endl;

    float* d_data;
    hipError_t error = hipMalloc(&d_data, 100*sizeof(float));
    CudaException::throw_if_error("hipMalloc");

    error = hipFree(d_data);
    CudaException::throw_if_error("hipFree");
    return 0;
}